#include "hip/hip_runtime.h"
/*
 * This file was created automatically from SUIF
 *   on Fri Nov  4 15:08:58 2011.
 */
#include <stdio.h>
//#include <cutil.h>

#define __suif_min(x,y) ((x)<(y)?(x):(y))

;
#define N 4096
extern void MV_GPU_wrapper(float (*)[4096], float *, float *);

extern int hipMemcpy();
extern int hipFree();
extern void __syncthreads();
extern int hipMemcpyToSymbol();
extern __global__ void mv_GPU(float *, float (*)[4096], float *);

int compare(float *a, float *b, int size, double threshold) {
  int i;
  int valid = 1;
  for (i=0; i<size; i++) {
    if (abs(a[i]-b[i]) > threshold) {
      printf("Got %.4f but expected %.4f at pos %d\n", b[i], a[i], i);
      valid = 0;
    }
  }
  return valid;
}

void normalMV(float *t, float* b, float *data, int* ptr, int* indices, int nr){
  int i, j;
  for (i=0; i<nr; i++) {                                                      
    for (j = ptr[i]; j<ptr[i+1]; j++) {
      //printf("Doing: t[%d] = t[%d] + data[%d] * b[indices[%d]]\n", i, i, j, j);
      t[i] = t[i] + data[j] * b[indices[j]];
    }
  }
}

extern void MV_GPU_wrapper(float *a, float *c, float *b) {
  return;
}

extern __global__ void mv_GPU(float *t, float* b, float *data, int* ptr, int* indices) {
  int bx = blockIdx.x;
  //int tx = threadIdx.x;
  //printf("bx: %d, tx: %d\n", bx, tx);
  
  int j;
  for (j = ptr[bx]; j<ptr[bx+1]; j++) {
    //printf("Doing: t[%d] = t[%d] + data[%d] * b[indices[%d]]\n", tx, tx, j, j);   
    t[bx] = t[bx] + data[j] * b[indices[j]];
  }

  return;
}

main (int argc, char **argv) {
  FILE *fp;
  char line[1024]; 
  int *ptr, *indices;
  float *data, *b, *t;
  int i;
  int n; // number of nonzero elements in data
  int nr; // number of rows in matrix
  int nc; // number of columns in matrix

  // Open input file and read to end of comments
  if (argc !=2) abort(); 

  if ((fp = fopen(argv[1], "r")) == NULL) {
    abort();
  }

  fgets(line, 128, fp);
  while (line[0] == '%') {
    fgets(line, 128, fp); 
  }

  // Read number of rows (nr), number of columns (nc) and
  // number of elements and allocate memory for ptr, indices, data, b and t.
  sscanf(line,"%d %d %d\n", &nr, &nc, &n);
  ptr = (int *) malloc ((nr+1)*sizeof(int));
  indices = (int *) malloc(n*sizeof(int));
  data = (float *) malloc(n*sizeof(float));
  b = (float *) malloc(nc*sizeof(float));
  t = (float *) malloc(nr*sizeof(float));

  // Read data in coordinate format and initialize sparse matrix
  int lastr=0;
  for (i=0; i<n; i++) {
    int r;
    fscanf(fp,"%d %d %f\n", &r, &(indices[i]), &(data[i]));  
    indices[i]--;  // start numbering at 0
    if (r!=lastr) { 
      ptr[r-1] = i; 
      lastr = r; 
    }
  }
  ptr[nr] = n;

  // initialize t to 0 and b with random data  
  for (i=0; i<nr; i++) {
    t[i] = 0.0;
  }
  for (i=0; i<nc; i++) {
    b[i] = (float) rand()/1111111111;
  }        

  // create CUDA event handles for timing purposes
  hipEvent_t start_event, stop_event;
  float elapsed_time_seq, elapsed_time_gpu;

  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);   
  normalMV(t, b, data, ptr, indices, nr);
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipEventElapsedTime(&elapsed_time_seq,start_event, stop_event);

  printf("Seq time: %.4f\n", elapsed_time_seq);


  float *tp, *bp, *datap, *result;
  int *ptrp, *indicesp;
  result = (float *) malloc(nr * sizeof(float));

  hipMalloc((void **)&tp, nr * 4);
  hipMalloc((void **)&bp, nc * 4);
  hipMemcpy(bp, b, nc * 4, hipMemcpyHostToDevice);
  hipMalloc((void **)&datap, n * 4);
  hipMemcpy(datap, data, n * 4, hipMemcpyHostToDevice);
  hipMalloc((void **)&ptrp, (nr+1) * 4);
  hipMemcpy(ptrp, ptr, (nr+1) * 4, hipMemcpyHostToDevice);
  hipMalloc((void **)&indicesp, n * 4);
  hipMemcpy(indicesp, indices, n * 4, hipMemcpyHostToDevice);

  dim3 dimGrid(nr, 1);
  dim3 dimBlock(1, 1);

  hipEventCreate(&start_event);
  hipEventCreate(&stop_event);
  hipEventRecord(start_event, 0);   
  mv_GPU<<<dimGrid,dimBlock>>>(tp, bp, datap, ptrp, indicesp);
  hipEventRecord(stop_event, 0);
  hipEventSynchronize(stop_event);
  hipMemcpy(result, tp, nr * 4, hipMemcpyDeviceToHost);
  hipEventElapsedTime(&elapsed_time_gpu,start_event, stop_event);

  hipFree(tp);
  hipFree(bp);
  hipFree(datap);
  hipFree(ptrp);
  hipFree(indicesp);

  printf("Par time: %.4f\n", elapsed_time_gpu);
  printf("Speedup: %.4f\n", elapsed_time_seq/elapsed_time_gpu);

  int res = compare(t, result, nr, 0.01); //lower threshold than before to account for some rare errors
  if (res == 1) {
    printf("VALID!\n");
  }
  else printf("INVALID...\n");
}
